#include "hip/hip_runtime.h"
typedef unsigned int u32_t;

#include "md5.h"

extern "C" __global__ __launch_bounds__(128,1) 
void deti_coins_cuda_kernel_search(u32_t *deti_coins_storage_area, u32_t custom_word_1, u32_t custom_word_2)
{
    u32_t t, n, a, b, c, d, state[4], x[16], coin[13], hash[4];

    n = (u32_t)threadIdx.x + (u32_t)blockDim.x * (u32_t)blockIdx.x;

    coin[0] = 0x49455444u;
    coin[1] = 0x696E6320u;
    coin[2] = 0x20662020u;
    coin[3] = 0x20202020u;
    coin[4] = 0x20202020u;
    coin[5] = 0x20202020u;
    coin[6] = 0x20202020u;
    coin[7] = 0x20202020u;
    coin[8] = 0x20202020u;
    coin[9] = 0x20202020u;
    coin[10] = custom_word_1;
    coin[11] = custom_word_2;
    coin[12] = 0x0A202020u;

    coin[4] += (n % 64) << 0; n /= 64;
    coin[4] += (n % 64) << 8; n /= 64;
    coin[4] += (n % 64) << 16; n /= 64;
    coin[4] += (n % 64) << 24; n /= 64;

    for(n = 0; n < 64; n++)
    {
        // 
        // Compute MD5 hash
        // 
#       define C(c) ((c))
#       define ROTATE(x, n) (((x) << (n)) | ((x) >> (32 - (n))))
#       define DATA(idx) coin[idx]
#       define HASH(idx) hash[idx]
#       define STATE(idx) state[idx]
#       define X(idx) x[idx]

        CUSTOM_MD5_CODE();
        if(hash[3] == 0){
            u32_t n = atomicAdd(deti_coins_storage_area,13);
            if(n + 13 <= 1024){
                for(t = 0; t<=12; t++){
                    deti_coins_storage_area[n + t] = coin[t];
                }
            }
        }
    }
}